#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// 错误检查宏
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA Error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
            exit(1); \
        } \
    } while (0)

// 向量加法的 CUDA kernel
__global__ void vectorAdd(const float *A, const float *B, float *C, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

// 矩阵乘法的 CUDA kernel
__global__ void matrixMul(const float *A, const float *B, float *C, 
                         int numARows, int numAColumns, int numBColumns) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numARows && col < numBColumns) {
        float sum = 0.0f;
        for (int i = 0; i < numAColumns; i++) {
            sum += A[row * numAColumns + i] * B[i * numBColumns + col];
        }
        C[row * numBColumns + col] = sum;
    }
}

// 向量加法示例
void vectorAddExample() {
    printf("\n=== Vector Addition Example ===\n");
    
    const int N = 1000000;
    size_t size = N * sizeof(float);

    // 分配主机内存
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // 初始化输入数据
    for (int i = 0; i < N; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_A, size));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_B, size));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_C, size));

    // 将数据复制到设备
    CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    // 启动 kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // 检查kernel执行错误
    CHECK_CUDA_ERROR(hipGetLastError());
    
    // 将结果复制回主机
    CHECK_CUDA_ERROR(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    // 验证结果
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = max(maxError, abs(h_C[i] - (h_A[i] + h_B[i])));
    }
    printf("Max error: %f\n", maxError);

    // 清理
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
}

// 矩阵乘法示例
void matrixMulExample() {
    printf("\n=== Matrix Multiplication Example ===\n");
    
    const int numARows = 1000;
    const int numAColumns = 1000;
    const int numBRows = numAColumns;
    const int numBColumns = 1000;
    const int numCRows = numARows;
    const int numCColumns = numBColumns;

    size_t sizeA = numARows * numAColumns * sizeof(float);
    size_t sizeB = numBRows * numBColumns * sizeof(float);
    size_t sizeC = numCRows * numCColumns * sizeof(float);

    // 分配主机内存
    float *h_A = (float *)malloc(sizeA);
    float *h_B = (float *)malloc(sizeB);
    float *h_C = (float *)malloc(sizeC);

    // 初始化输入矩阵
    for (int i = 0; i < numARows * numAColumns; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
    }
    for (int i = 0; i < numBRows * numBColumns; i++) {
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_A, sizeA));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_B, sizeB));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_C, sizeC));

    // 将数据复制到设备
    CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice));

    // 设置 kernel 配置
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((numCColumns + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (numCRows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 启动 kernel
    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C,
                                                 numARows, numAColumns,
                                                 numBColumns);

    // 检查kernel执行错误
    CHECK_CUDA_ERROR(hipGetLastError());

    // 将结果复制回主机
    CHECK_CUDA_ERROR(hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost));

    printf("Matrix multiplication completed\n");

    // 清理
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
}

int main() {
    // 打印 CUDA 设备信息
    hipDeviceProp_t prop;
    CHECK_CUDA_ERROR(hipGetDeviceProperties(&prop, 0));
    printf("Device: %s\n", prop.name);
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    
    // 运行示例
    vectorAddExample();
    matrixMulExample();
    
    return 0;
} 